#include "hip/hip_runtime.h"
#include <cusp/csr_matrix.h>
#include <cusp/print.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/bicg.h>
#include <cusp/krylov/bicgstab.h>
#include <cusp/krylov/gmres.h>
#include <cusp/transpose.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>

#include <stdio.h>
#include <hipsparse.h>
#include <hipblas.h>

#include <sys/time.h>

using namespace std;

#ifdef SINGLE
  typedef float real2;
#else
  typedef double real2;
#endif

extern int *rowStartsSmall, *colSmall, NN, NNZ, solverIterMax;
extern double solverTol;
extern real2 *uDiagonal, *vDiagonal, *wDiagonal, *u, *v, *w;
extern real2 *Cx, *Cy, *Cz;
extern real2 *F, *pPrime;
extern int *rowStartsDiagonal, *colDiagonal;

double getHighResolutionTime();

//-----------------------------------------------------------------------------
void CUSP_pC_CUDA_CR()
//-----------------------------------------------------------------------------
{

   double Start6, End6, Start7, End7;
   
   Start6 = getHighResolutionTime();         
   //---------------------------------------------- 
   //calculate arrays for x dimension
   //---------------------------------------------- 

   Start7 = getHighResolutionTime();   
   //---------------------------------------------- 
   // Copy C_x from host to device
   // Allocate stifness matrix C_x in CSR format
   cusp::csr_matrix<int, real2, cusp::device_memory> CCx(NN, NN, NNZ);
   thrust::copy(rowStartsSmall,rowStartsSmall + NN + 1,CCx.row_offsets.begin());
   thrust::copy(colSmall,colSmall +  NNZ,CCx.column_indices.begin());
   thrust::copy(Cx,Cx + NNZ,CCx.values.begin());
   //---------------------------------------------- 
   
   //---------------------------------------------- 
   // transpose(C_x)
   cusp::csr_matrix<int, real2, cusp::device_memory> CxT;
   cusp::transpose(CCx, CxT);
   //---------------------------------------------- 
   End7 = getHighResolutionTime();    
   printf("         Time for transpose(C_x)             = %-.4g seconds.\n", End7 - Start7);       

   Start7 = getHighResolutionTime();     
   //---------------------------------------------- 
   // Copy K_u^(-1) from host to device 
   // Allocate stifness matrix K_u^(-1) in CSR format   
   cusp::csr_matrix<int, real2, cusp::device_memory> uDiagonal_CUSP(NN, NN, NN);
   thrust::copy(rowStartsDiagonal,rowStartsDiagonal + NN + 1,uDiagonal_CUSP.row_offsets.begin());
   thrust::copy(colDiagonal,colDiagonal +  NN,uDiagonal_CUSP.column_indices.begin());
   thrust::copy(uDiagonal,uDiagonal + NN,uDiagonal_CUSP.values.begin()); 
   //----------------------------------------------     

   //---------------------------------------------- 
   // Copy velocities from host to device memory
   cusp::array1d<real2, cusp::device_memory> u_CUSP(NN);   
   thrust::copy(u, u + NN, u_CUSP.begin()); 
   //----------------------------------------------
   
   //----------------------------------------------  
   // RHS of the equation [4a]  
   // transpose(C_x)*u 
   // \______________/ 
   //        F1             
   cusp::array1d<real2, cusp::device_memory> F1(NN);
   cusp::multiply(CxT, u_CUSP, F1);   
   cusp::array1d<real2, cusp::device_memory> Fsum(NN);
   cusp::blas::fill(Fsum,0.0);   
   cusp::blas::axpy(F1,Fsum,-1); 
   //---------------------------------------------- 
   
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      F1.swap(tmp);
   }    
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      u_CUSP.swap(tmp);
   }  
   End7 = getHighResolutionTime();    
   printf("         Time for sum [transpose(C_x)*u]     = %-.4g seconds.\n", End7 - Start7);    

   Start7 = getHighResolutionTime();       
   //---------------------------------------------- 
   // LHS of the equation [4a]
   // transpose(C_x)*(diagonal(K_u)^-1
   cusp::csr_matrix<int, real2, cusp::device_memory> CxTdia;
   cusp::multiply(CxT, uDiagonal_CUSP, CxTdia);
   //----------------------------------------------    
   
   {
      // create temporary empty matrix to delete array
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      CxT.swap(tmp);
   } 
   {
      // create temporary empty matrix
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      uDiagonal_CUSP.swap(tmp);
   }   
   End7 = getHighResolutionTime();   
   printf("         Time for [transpose(C_x)] * K_u     = %-.4g seconds.\n", End7 - Start7);  
   
   Start7 = getHighResolutionTime();    
   //----------------------------------------------   
   // LHS of the equation [4a]
   // [transpose(C_x)*(diagonal(K_u)^-1]*C_x 
   // \________________________________/
   //          from above (CxTdia)   
   cusp::csr_matrix<int, real2, cusp::device_memory> valx;   
   cusp::multiply(CxTdia, CCx, valx);   
   //----------------------------------------------    
   End7 = getHighResolutionTime();   
   printf("         Time for [trans(C_x)*K_u] * C_x     = %-.4g seconds.\n", End7 - Start7);     
   End6 = getHighResolutionTime();   
   printf("      Time for calc pC arrays for x dim   = %-.4g seconds.\n", End6 - Start6); 
   
   
   Start6 = getHighResolutionTime();      
   //---------------------------------------------- 
   //calculate arrays for y dimension
   //---------------------------------------------- 

   Start7 = getHighResolutionTime();   
   //---------------------------------------------- 
   // Copy C_y from host to device
   // Allocate stifness matrix C_y in CSR format
   cusp::csr_matrix<int, real2, cusp::device_memory> CCy(NN, NN, NNZ);
   thrust::copy(rowStartsSmall,rowStartsSmall + NN + 1,CCy.row_offsets.begin());
   thrust::copy(colSmall,colSmall +  NNZ,CCy.column_indices.begin());
   thrust::copy(Cy,Cy + NNZ,CCy.values.begin());
   //---------------------------------------------- 
   
   //---------------------------------------------- 
   // transpose(C_y)
   cusp::csr_matrix<int, real2, cusp::device_memory> CyT;
   cusp::transpose(CCy, CyT);
   //----------------------------------------------   
   End7 = getHighResolutionTime();    
   printf("         Time for transpose(C_y)             = %-.4g seconds.\n", End7 - Start7);   
      
   Start7 = getHighResolutionTime();         
   //---------------------------------------------- 
   // Copy K_v^(-1) from host to device 
   // Allocate stifness matrix K_v^(-1) in CSR format   
   cusp::csr_matrix<int, real2, cusp::device_memory> vDiagonal_CUSP(NN, NN, NN);
   thrust::copy(rowStartsDiagonal,rowStartsDiagonal + NN + 1,vDiagonal_CUSP.row_offsets.begin());
   thrust::copy(colDiagonal,colDiagonal +  NN,vDiagonal_CUSP.column_indices.begin());
   thrust::copy(vDiagonal,vDiagonal + NN,vDiagonal_CUSP.values.begin()); 
   //----------------------------------------------     

   //---------------------------------------------- 
   // Copy velocities from host to device memory
   cusp::array1d<real2, cusp::device_memory> v_CUSP(NN);   
   thrust::copy(v, v + NN, v_CUSP.begin()); 
   //----------------------------------------------
   
   //----------------------------------------------  
   // RHS of the equation [4a]  
   // transpose(C_y)*u 
   // \______________/ 
   //        F2             
   cusp::array1d<real2, cusp::device_memory> F2(NN);
   cusp::multiply(CyT, v_CUSP, F2);     
   cusp::blas::axpy(F2,Fsum,-1); 
   //---------------------------------------------- 
   
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      F2.swap(tmp);
   }    
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      v_CUSP.swap(tmp);
   }       
   End7 = getHighResolutionTime();    
   printf("         Time for sum [transpose(C_y)*v]     = %-.4g seconds.\n", End7 - Start7);
   
   Start7 = getHighResolutionTime();      
   //---------------------------------------------- 
   // LHS of the equation [4a]
   // transpose(C_y)*(diagonal(K_v)^-1
   cusp::csr_matrix<int, real2, cusp::device_memory> CyTdia;
   cusp::multiply(CyT, vDiagonal_CUSP, CyTdia);
   //----------------------------------------------    
   
   {
      // create temporary empty matrix to delete array
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      CyT.swap(tmp);
   } 
   {
      // create temporary empty matrix
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      vDiagonal_CUSP.swap(tmp);
   }
   End7 = getHighResolutionTime();   
   printf("         Time for [transpose(C_y)] * K_v     = %-.4g seconds.\n", End7 - Start7);   

   Start7 = getHighResolutionTime();   
   //----------------------------------------------   
   // LHS of the equation [4a]
   // [transpose(C_y)*(diagonal(K_v)^-1]*C_y 
   // \________________________________/
   //          from above (CyTdia)   
   cusp::csr_matrix<int, real2, cusp::device_memory> valy;   
   cusp::multiply(CyTdia, CCy, valy);   
   // summing x, y components
   // [transpose(C_x)*(diagonal(K_u)^-1]*C_x + [transpose(C_y)*(diagonal(K_v)^-1]*C_y
   cusp::blas::axpy(valy.values,valx.values,1);
   //----------------------------------------------  
   
   {
      // create temporary empty matrix
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      valy.swap(tmp);
   }  
   End7 = getHighResolutionTime();   
   printf("         Time for [trans(C_y)*K_v] * C_y     = %-.4g seconds.\n", End7 - Start7);     
   End6 = getHighResolutionTime();   
   printf("      Time for calc pC arrays for y dim   = %-.4g seconds.\n", End6 - Start6);    
   

   Start6 = getHighResolutionTime();    
   //---------------------------------------------- 
   //calculate arrays for z dimension
   //---------------------------------------------- 
   
   Start7 = getHighResolutionTime();   
   //---------------------------------------------- 
   // Copy C_z from host to device
   // Allocate stifness matrix C_y in CSR format
   cusp::csr_matrix<int, real2, cusp::device_memory> CCz(NN, NN, NNZ);
   thrust::copy(rowStartsSmall,rowStartsSmall + NN + 1,CCz.row_offsets.begin());
   thrust::copy(colSmall,colSmall +  NNZ,CCz.column_indices.begin());
   thrust::copy(Cz,Cz + NNZ,CCz.values.begin());
   //---------------------------------------------- 
   
   //---------------------------------------------- 
   // transpose(C_z)
   cusp::csr_matrix<int, real2, cusp::device_memory> CzT;
   cusp::transpose(CCz, CzT);
   //----------------------------------------------
   End7 = getHighResolutionTime();    
   printf("         Time for transpose(C_z)             = %-.4g seconds.\n", End7 - Start7);   
      
   Start7 = getHighResolutionTime();      
   //---------------------------------------------- 
   // Copy K_w^(-1) from host to device 
   // Allocate stifness matrix K_w^(-1) in CSR format   
   cusp::csr_matrix<int, real2, cusp::device_memory> wDiagonal_CUSP(NN, NN, NN);
   thrust::copy(rowStartsDiagonal,rowStartsDiagonal + NN + 1,wDiagonal_CUSP.row_offsets.begin());
   thrust::copy(colDiagonal,colDiagonal +  NN,wDiagonal_CUSP.column_indices.begin());
   thrust::copy(wDiagonal,wDiagonal + NN,wDiagonal_CUSP.values.begin()); 
   //----------------------------------------------     

   //---------------------------------------------- 
   // Copy velocities from host to device memory
   cusp::array1d<real2, cusp::device_memory> w_CUSP(NN);   
   thrust::copy(w, w + NN, w_CUSP.begin()); 
   //----------------------------------------------
   
   //----------------------------------------------  
   // RHS of the equation [4a]  
   // transpose(C_z)*u 
   // \______________/ 
   //        F2             
   cusp::array1d<real2, cusp::device_memory> F3(NN);
   cusp::multiply(CzT, w_CUSP, F3);     
   cusp::blas::axpy(F3,Fsum,-1); 
   //---------------------------------------------- 
   
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      F3.swap(tmp);
   }    
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      w_CUSP.swap(tmp);
   }
   End7 = getHighResolutionTime();    
   printf("         Time for sum [transpose(C_z)*w]     = %-.4g seconds.\n", End7 - Start7);   

   Start7 = getHighResolutionTime();   
   //---------------------------------------------- 
   // LHS of the equation [4a]
   // transpose(C_z)*(diagonal(K_w)^-1
   cusp::csr_matrix<int, real2, cusp::device_memory> CzTdia;
   cusp::multiply(CzT, wDiagonal_CUSP, CzTdia);
   //----------------------------------------------    
   
   {
      // create temporary empty matrix to delete array
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      CzT.swap(tmp);
   } 
   {
      // create temporary empty matrix to delete array
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      wDiagonal_CUSP.swap(tmp);
   }   
   End7 = getHighResolutionTime();   
   printf("         Time for [transpose(C_z)] * K_w     = %-.4g seconds.\n", End7 - Start7);   

   Start7 = getHighResolutionTime();   
   //----------------------------------------------   
   // LHS of the equation [4a]
   // [transpose(C_z)*(diagonal(K_w)^-1]*C_z 
   // \________________________________/
   //          from above (CzTdia)   
   cusp::csr_matrix<int, real2, cusp::device_memory> valz;   
   cusp::multiply(CzTdia, CCz, valz);   
   // summing x, y, z components
   // [transpose(C_x)*(diagonal(K_u)^-1]*C_x + [transpose(C_y)*(diagonal(K_v)^-1]*C_y + [transpose(C_z)*(diagonal(K_w)^-1]*C_z
   cusp::blas::axpy(valz.values,valx.values,1);
   //----------------------------------------------  
   
   {
      // create temporary empty matrix
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      valz.swap(tmp);
   }  
   End7 = getHighResolutionTime();   
   printf("         Time for [trans(C_z)*K_w] * C_z     = %-.4g seconds.\n", End7 - Start7);   
   End6 = getHighResolutionTime();   
   printf("      Time for calc pC arrays for z dim   = %-.4g seconds.\n", End6 - Start6); 
   
   
   Start6 = getHighResolutionTime();       
   // Copy resulting LHS and RHS vectors from device memory to host memory
   int *row_deltaP, *col_deltaP;
   real2 *val_deltaP, *F_deltaP;
   
   row_deltaP = new int[NN+1];
   col_deltaP = new int[valx.row_offsets[NN]];
   val_deltaP = new real2[valx.row_offsets[NN]];

   thrust::copy(valx.row_offsets.begin(), valx.row_offsets.end(), row_deltaP);
   thrust::copy(valx.column_indices.begin(), valx.column_indices.end(), col_deltaP);
   thrust::copy(valx.values.begin(), valx.values.end(), val_deltaP);
   {
      // create temporary empty matrix to delete array
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      valx.swap(tmp);
   }  
   
   F_deltaP = new real2[NN];
   thrust::copy(Fsum.begin(), Fsum.end(), F_deltaP);  
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      Fsum.swap(tmp);
   }    
   
   
   //----------------------------------------------
   //-------------CONJUGATE RESIDUAL---------------
   
   real2 a, b, r0, r1, residual;
   int k;
   int *d_col, *d_row;
   real2 *d_val, *d_x, *d_r, *d_p, *d_Ax, *d_Ar;  
   
   hipsparseHandle_t handle = 0;
   hipsparseStatus_t status;
   status = hipsparseCreate(&handle);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      fprintf( stderr, "!!!! CUSPARSE initialization error\n" );
   }
   hipsparseMatDescr_t descr = 0;
   status = hipsparseCreateMatDescr(&descr); 
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      fprintf( stderr, "!!!! CUSPARSE hipsparseCreateMatDescr error\n" );
   } 

   hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
   hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
   
   hipMalloc((void**)&d_col, row_deltaP[NN]*sizeof(int)) ;
   hipMalloc((void**)&d_row, (NN+1)*sizeof(int)) ;
   hipMalloc((void**)&d_val, row_deltaP[NN]*sizeof(real2)) ;
   hipMalloc((void**)&d_x, NN*sizeof(real2)) ;  
   hipMalloc((void**)&d_r, NN*sizeof(real2)) ;
   hipMalloc((void**)&d_p, NN*sizeof(real2)) ;
   hipMalloc((void**)&d_Ax, NN*sizeof(real2)) ;
   hipMalloc((void**)&d_Ar, NN*sizeof(real2)) ;  

   hipMemcpy(d_col, col_deltaP, row_deltaP[NN]*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(d_row, row_deltaP, (NN+1)*sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(d_val, val_deltaP, row_deltaP[NN]*sizeof(real2), hipMemcpyHostToDevice);
   hipMemcpy(d_x, pPrime, NN*sizeof(real2), hipMemcpyHostToDevice);
   hipMemcpy(d_r, F_deltaP, NN*sizeof(real2), hipMemcpyHostToDevice);
   
   delete[] col_deltaP;
   delete[] row_deltaP;
   delete[] val_deltaP;
   delete[] F_deltaP;
   End6 = getHighResolutionTime();   
   printf("      Time for init variables for CR      = %-.4g seconds.\n", End6 - Start6); 

   Start6 = getHighResolutionTime();    
   hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, NN, NN, 1.0, descr, d_val, d_row, d_col, d_x, 0.0, d_Ax);
   // y = alpha * op(A) * x + beta * y
   // hipsparseDcsrmv(handle,�hipsparseOperation_t�transA, m,�n,�alpha, descrA, *csrValA, *csrRowPtrA, *csrColIndA, *x,�beta, *y�)
   // descrA = matrix property of A
   // d_Ax = 1.0 * A * d_x + 0.0 * d_Ax
   
   hipblasDaxpy(NN, -1.0, d_Ax, 1, d_r, 1);
   // hipblasDaxpy(int n, *alpha, *x, incx, *y, incy)
   // y[j] = alpha * x[k] + y[j]
   // d_r = -1.0 * d_Ax + d_r

   hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, NN, NN, 1.0 , descr, d_val, d_row, d_col, d_r, 0.0, d_Ar);
   // y = alpha * op(A) * x + beta * y
   // hipsparseDcsrmv(handle,�hipsparseOperation_t�transA, m,�n,�alpha, descrA, *csrValA, *csrRowPtrA, *csrColIndA, *x,�beta, *y�)
   // descrA = matrix property of A  
   // d_Ar = 1.0 * A * d_r + 0.0 * d_Ar
      
   r1 = hipblasDdot(NN, d_r, 1, d_Ar, 1);
   // result = hipblasDdot(int n, *x, incx, *y, incy)
   // result = total(x[k] � y[j]) 
   // r1 = total(d_r[i] * d_Ar[i])
   
   residual = hipblasDdot(NN, d_r, 1, d_r, 1);
   // result = hipblasDdot(int n, *x, incx, *y, incy)
   // result = total(x[k] � y[j]) 
   // residual = total(d_r[i] * d_r[i])
   
   k = 1;
   while (residual > solverTol*solverTol && k <= solverIterMax) {
      if (k > 1) {
         b = r1 / r0;
 
         hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, NN, NN, b, descr, d_val, d_row, d_col, d_p, 0.0, d_Ax);
         // y = alpha * op(A) * x + beta * y
         // hipsparseDcsrmv(handle,�hipsparseOperation_t�transA, m,�n,�alpha, descrA, *csrValA, *csrRowPtrA, *csrColIndA, *x,�beta, *y�)
         // descrA = matrix property of A  
         // d_Ax = b * A * d_p + 0.0 * d_Ax 
         
         hipblasDaxpy(NN, 1.0, d_Ar, 1, d_Ax, 1);
         // hipblasDaxpy(int n, *alpha, *x, incx, *y, incy)
         // y[j] = alpha * x[k] + y[j]
         // d_Ax = 1.0 * d_Ar + d_Ax           
         
         hipblasDscal(NN, b, d_p, 1);
         // hipblasDscal(int n, *alpha, *x, incx)
         // x[j] = alpha * x[j]
         // d_p = b * d_p         
         
         hipblasDaxpy(NN, 1.0, d_r, 1, d_p, 1);
         // hipblasDaxpy(int n, *alpha, *x, incx, *y, incy)
         // y[j] = alpha * x[k] + y[j]
         // d_p = 1.0 * d_r + d_p           
           
      } else {
         hipblasDcopy(NN, d_r, 1, d_p, 1);
         // d_p = d_r
         
         hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, NN, NN, 1.0, descr, d_val, d_row, d_col, d_p, 0.0, d_Ax);
         // y = alpha * op(A) * x + beta * y
         // hipsparseDcsrmv(handle,�hipsparseOperation_t�transA, m,�n,�alpha, descrA, *csrValA, *csrRowPtrA, *csrColIndA, *x,�beta, *y�)
         // descrA = matrix property of A  
         // d_Ax = 1.0 * A * d_p + 0.0 * d_Ax         
      }
      
      a = r1 / hipblasDdot(NN, d_Ax, 1, d_Ax, 1);
      // a = r1 / total(d_Ax[i] * d_Ax[i]) 
      
      hipblasDaxpy(NN, a, d_p, 1, d_x, 1);
      // hipblasDaxpy(int n, *alpha, *x, incx, *y, incy)
      // y[j] = alpha * x[k] + y[j]
      // d_x = a * d_p + d_x   
      
      hipblasDaxpy(NN, -a, d_Ax, 1, d_r, 1);
      // hipblasDaxpy(int n, *alpha, *x, incx, *y, incy)
      // y[j] = alpha * x[k] + y[j]
      // d_r = -a * d_Ax + d_r      

      r0 = r1;
      
      hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, NN, NN, 1.0 , descr, d_val, d_row, d_col, d_r, 0.0, d_Ar);
      // y = alpha * op(A) * x + beta * y
      // hipsparseDcsrmv(handle,�hipsparseOperation_t�transA, m,�n,�alpha, descrA, *csrValA, *csrRowPtrA, *csrColIndA, *x,�beta, *y�)
      // descrA = matrix property of A  
      // d_Ar = 1.0 * A * d_r + 0.0 * d_Ar
      
      r1 = hipblasDdot(NN, d_r, 1, d_Ar, 1);
      // result = hipblasDdot(int n, *x, incx, *y, incy)
      // result = total(x[k] � y[j]) 
      // r1 = total(d_r[i] * d_Ar[i])  
      
      hipDeviceSynchronize();
      k++;
      
      residual = hipblasDdot(NN, d_r, 1, d_r, 1);
      // result = hipblasDdot(int n, *x, incx, *y, incy)
      // result = total(x[k] � y[j]) 
      // residual = total(d_r[i] * d_r[i])      
   }

   hipMemcpy(pPrime, d_x, NN*sizeof(real2), hipMemcpyDeviceToHost);
   End6 = getHighResolutionTime();   
   printf("      Time for CR calculations            = %-.4g seconds.\n", End6 - Start6);      
   
   hipsparseDestroy(handle);
   hipFree(d_col);
   hipFree(d_row);
   hipFree(d_val);
   hipFree(d_x);
   hipFree(d_r);
   hipFree(d_p);
   hipFree(d_Ax);   
   hipFree(d_Ar);
   
   if (k > solverIterMax) {
      std::cout << "      Solver reached iteration limit " << k-1 << " before converging";      
      std::cout << " to " <<  solverTol ;
      std::cout << ", final residual is " << sqrt(residual) << endl;
   }
   else {
      std::cout << "      Solver converged to " << sqrt(residual) << " relative tolerance";
      std::cout << " after " << k-1 << " iterations" << endl;
   }

}  // End of function CUSP_pC_CUDA_CR()
