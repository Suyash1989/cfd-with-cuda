#include <stdio.h>
#include <iostream>
#include <hipsparse.h>
#include <hipblas.h>

using namespace std;

#ifdef SINGLE
  typedef float real2;
#else
  typedef double real2;
#endif

extern int   *rowStarts, *col, NN, solverIterMax, solverIter, bigNumber;
extern double solverTol, solverNorm;
extern real2 *u, *val, *F;
time_t start, end;




//-------------------------------------------------------------------------
void CUSPARSEsolver()
//-------------------------------------------------------------------------
{
   int *d_col, *d_row;
   real2 a, b, r0, r1;
   real2 *d_val, *d_x;
   real2 *d_r, *d_p, *d_Ax;

   int i, k;
   real2 *val_real2, *F_real2;
   
   //-------------------------------------------------------------------------------
   // Converting val and F values from double to real2 
   val_real2 = new real2[rowStarts[NN]];
   for(i=0; i<rowStarts[NN]; i++) {
      val_real2[i] = real2(val[i]);
   }

   F_real2 = new real2[NN];
   for(i=0; i<NN; i++) {
      F_real2[i] = real2(F[i])*bigNumber;
   }
   //------------------------------------------------------------------------------- 

   hipsparseHandle_t handle = 0;
   hipsparseStatus_t status;
   status = hipsparseCreate(&handle);
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      fprintf( stderr, "!!!! CUSPARSE initialization error\n" );
   }
   hipsparseMatDescr_t descr = 0;
   status = hipsparseCreateMatDescr(&descr); 
   if (status != HIPSPARSE_STATUS_SUCCESS) {
      fprintf( stderr, "!!!! CUSPARSE hipsparseCreateMatDescr error\n" );
   } 

   hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
   hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
   
   for(i=0; i<NN; i++) {
      u[i] = 0.0;
   }

   hipMalloc((void**)&d_col, (rowStarts[NN])*sizeof(int));
   hipMalloc((void**)&d_row, (NN+1)*sizeof(int));
   hipMalloc((void**)&d_val, (rowStarts[NN])*sizeof(real2));
   hipMalloc((void**)&d_x,  NN*sizeof(real2));  
   hipMalloc((void**)&d_r, NN*sizeof(real2));
   hipMalloc((void**)&d_p, NN*sizeof(real2));
   hipMalloc((void**)&d_Ax, NN*sizeof(real2));

   hipMemcpy(d_col, col, (rowStarts[NN])*sizeof(int), hipMemcpyHostToDevice);  
   hipMemcpy(d_row, rowStarts, (NN+1)*sizeof(int), hipMemcpyHostToDevice);  
   hipMemcpy(d_val, val_real2, (rowStarts[NN])*sizeof(real2), hipMemcpyHostToDevice); 
   hipMemcpy(d_x, u, NN*sizeof(real2), hipMemcpyHostToDevice);
   hipMemcpy(d_r, F_real2, NN*sizeof(real2), hipMemcpyHostToDevice);

   #ifdef SINGLE
      hipsparseScsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, NN, NN, 1.0, descr, d_val, d_row, d_col, d_x, 0.0, d_Ax);
      hipblasSaxpy(NN, -1.0, d_Ax, 1, d_r, 1);
      r1 = hipblasSdot(NN, d_r, 1, d_r, 1);
   #else
      hipsparseDcsrmv(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE, NN, NN, 1.0, descr, d_val, d_row, d_col, d_x, 0.0, d_Ax);
      hipblasDaxpy(NN, -1.0, d_Ax, 1, d_r, 1);
      r1 = hipblasDdot(NN, d_r, 1, d_r, 1);
   #endif
   r0=0;
   
   k = 1;
   while (r1 > solverTol*solverTol && k <= solverIterMax) {
      if (k > 1) {
         b = r1 / r0;
         #ifdef SINGLE
            hipblasSscal(NN, b, d_p, 1);
            hipblasSaxpy(NN, 1.0, d_r, 1, d_p, 1);
         #else
            hipblasDscal(NN, b, d_p, 1);
            hipblasDaxpy(NN, 1.0, d_r, 1, d_p, 1);
         #endif
      } else {
         #ifdef SINGLE
            hipblasScopy(NN, d_r, 1, d_p, 1);
         #else
            hipblasDcopy(NN, d_r, 1, d_p, 1);
         #endif
      }

      #ifdef SINGLE
         hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, NN, NN, 1.0, descr, d_val, d_row, d_col, d_p, 0.0, d_Ax);
         a = r1 / hipblasSdot(NN, d_p, 1, d_Ax, 1);
         hipblasSaxpy(NN, a, d_p, 1, d_x, 1);
         hipblasSaxpy(NN, -a, d_Ax, 1, d_r, 1);

         r0 = r1;
         r1 = hipblasSdot(NN, d_r, 1, d_r, 1);
      #else
         hipsparseDcsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, NN, NN, 1.0, descr, d_val, d_row, d_col, d_p, 0.0, d_Ax);
         a = r1 / hipblasDdot(NN, d_p, 1, d_Ax, 1);
         hipblasDaxpy(NN, a, d_p, 1, d_x, 1);
         hipblasDaxpy(NN, -a, d_Ax, 1, d_r, 1);

         r0 = r1;
         r1 = hipblasDdot(NN, d_r, 1, d_r, 1);
      #endif

      hipDeviceSynchronize();
      k++;
   }
   
   //-------------------------------------------------------------------------------
   // Writes CG solution answers
   hipMemcpy(u, d_x, (NN)*sizeof(real2), hipMemcpyDeviceToHost);
   cout << endl;
   //cout << endl;
   //for(i=0; i<NN; i++) {
   //   printf("%f \n", u[i]);
   //}
   //cout << endl;
   cout <<"number of iterations: "<< k << endl;
   //-------------------------------------------------------------------------------

   hipsparseDestroy(handle);

   hipFree(d_col);
   hipFree(d_row);
   hipFree(d_val);
   hipFree(d_x);
   hipFree(d_r);
   hipFree(d_p);
   hipFree(d_Ax);
}

