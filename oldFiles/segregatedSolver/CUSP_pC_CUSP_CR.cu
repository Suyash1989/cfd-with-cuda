#include <cusp/csr_matrix.h>
#include <cusp/print.h>
#include <cusp/precond/diagonal.h>
#include <cusp/relaxation/jacobi.h>
#include <cusp/relaxation/polynomial.h>
#include <cusp/precond/aggregation/smoothed_aggregation.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/cr.h>
#include <cusp/krylov/bicg.h>
#include <cusp/krylov/bicgstab.h>
#include <cusp/krylov/gmres.h>
#include <cusp/transpose.h>
#include <cusp/array2d.h>
#include <cusp/multiply.h>
#include <cusp/blas.h>

#include <stdio.h>
#include <hipsparse.h>
#include <hipblas.h>

#include <sys/time.h>

using namespace std;

#ifdef SINGLE
  typedef float real2;
#else
  typedef double real2;
#endif

extern int *rowStartsSmall, *colSmall, NN, NNZ, solverIterMax;
extern double solverTol;
extern real2 *K_u_diagonal, *K_v_diagonal, *K_w_diagonal, *u, *v, *w;
extern real2 *Cx, *Cy, *Cz;
extern real2 *CxT, *CyT, *CzT;
extern real2 *delta_p;
extern int *rowStartsDiagonal, *colDiagonal;
extern real2 *F_deltaP, *val_deltaP;
extern int *row_deltaP, *col_deltaP;
extern int iter;

void applyBC_deltaP();
double getHighResolutionTime();

//-----------------------------------------------------------------------------
void CUSP_pC_CUSP_CR()
//-----------------------------------------------------------------------------
{

   double Start6, End6, Start7, End7;
   
   Start6 = getHighResolutionTime();         
   //---------------------------------------------- 
   //calculate arrays for x dimension
   //---------------------------------------------- 

   Start7 = getHighResolutionTime();   
   //---------------------------------------------- 
   // Copy C_x from host to device
   // Allocate stifness matrix C_x in CSR format
   cusp::csr_matrix<int, real2, cusp::device_memory> Cx_CUSP(NN, NN, NNZ);
   thrust::copy(rowStartsSmall,rowStartsSmall + NN + 1,Cx_CUSP.row_offsets.begin());
   thrust::copy(colSmall,colSmall +  NNZ,Cx_CUSP.column_indices.begin());
   thrust::copy(Cx,Cx + NNZ,Cx_CUSP.values.begin());
   //---------------------------------------------- 
   
   //---------------------------------------------- 
   // Copy transpose(C_x) from host to device
   // Allocate stifness matrix transpose(C_x) in CSR format
   cusp::csr_matrix<int, real2, cusp::device_memory> CxT_CUSP(NN, NN, NNZ);
   thrust::copy(rowStartsSmall,rowStartsSmall + NN + 1,CxT_CUSP.row_offsets.begin());
   thrust::copy(colSmall,colSmall +  NNZ,CxT_CUSP.column_indices.begin());
   thrust::copy(CxT,CxT + NNZ,CxT_CUSP.values.begin());
   //---------------------------------------------- 
   End7 = getHighResolutionTime();    
   printf("         Time for copy Cx and CxT            = %-.4g seconds.\n", End7 - Start7);       

   Start7 = getHighResolutionTime();     
   //---------------------------------------------- 
   // Copy K_u^(-1) from host to device 
   // Allocate stifness matrix K_u^(-1) in CSR format   
   cusp::csr_matrix<int, real2, cusp::device_memory> K_u_diagonal_CUSP(NN, NN, NN);
   thrust::copy(rowStartsDiagonal,rowStartsDiagonal + NN + 1,K_u_diagonal_CUSP.row_offsets.begin());
   thrust::copy(colDiagonal,colDiagonal +  NN,K_u_diagonal_CUSP.column_indices.begin());
   thrust::copy(K_u_diagonal,K_u_diagonal + NN,K_u_diagonal_CUSP.values.begin()); 
   //----------------------------------------------     

   //---------------------------------------------- 
   // Copy velocities from host to device memory
   cusp::array1d<real2, cusp::device_memory> u_CUSP(NN);   
   thrust::copy(u, u + NN, u_CUSP.begin()); 
   //----------------------------------------------
   
   //----------------------------------------------  
   // RHS of the equation [4a]  
   // transpose(C_x)*u 
   // \______________/ 
   //        F1             
   cusp::array1d<real2, cusp::device_memory> F1(NN);
   cusp::multiply(CxT_CUSP, u_CUSP, F1);   
   cusp::array1d<real2, cusp::device_memory> Fsum(NN);
   cusp::blas::fill(Fsum,0.0);   
   cusp::blas::axpy(F1,Fsum,-1); 
   //---------------------------------------------- 
   
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      F1.swap(tmp);
   }    
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      u_CUSP.swap(tmp);
   }  
   End7 = getHighResolutionTime();    
   printf("         Time for sum [transpose(C_x)*u]     = %-.4g seconds.\n", End7 - Start7);    

   Start7 = getHighResolutionTime();       
   //---------------------------------------------- 
   // LHS of the equation [4a]
   // transpose(C_x)*(diagonal(K_u)^-1
   cusp::csr_matrix<int, real2, cusp::device_memory> CxTdia;
   cusp::multiply(CxT_CUSP, K_u_diagonal_CUSP, CxTdia);
   //----------------------------------------------    
   
   {
      // create temporary empty matrix to delete array
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      CxT_CUSP.swap(tmp);
   } 
   {
      // create temporary empty matrix
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      K_u_diagonal_CUSP.swap(tmp);
   }   
   End7 = getHighResolutionTime();   
   printf("         Time for [transpose(C_x)] * K_u     = %-.4g seconds.\n", End7 - Start7);  
   
   Start7 = getHighResolutionTime();    
   //----------------------------------------------   
   // LHS of the equation [4a]
   // [transpose(C_x)*(diagonal(K_u)^-1]*C_x 
   // \________________________________/
   //          from above (CxTdia)   
   cusp::csr_matrix<int, real2, cusp::device_memory> valx;   
   cusp::multiply(CxTdia, Cx_CUSP, valx);   
   // cout << "NNZ K pressure correction = " << valx.row_offsets[NN] << endl;
   //----------------------------------------------    
   End7 = getHighResolutionTime();   
   printf("         Time for [trans(C_x)*K_u] * C_x     = %-.4g seconds.\n", End7 - Start7);     
   End6 = getHighResolutionTime();   
   printf("      Time for calc pC arrays for x dim   = %-.4g seconds.\n", End6 - Start6); 
   
   
   Start6 = getHighResolutionTime();      
   //---------------------------------------------- 
   //calculate arrays for y dimension
   //---------------------------------------------- 

   Start7 = getHighResolutionTime();   
   //---------------------------------------------- 
   // Copy C_y from host to device
   // Allocate stifness matrix C_y in CSR format
   cusp::csr_matrix<int, real2, cusp::device_memory> Cy_CUSP(NN, NN, NNZ);
   thrust::copy(rowStartsSmall,rowStartsSmall + NN + 1,Cy_CUSP.row_offsets.begin());
   thrust::copy(colSmall,colSmall +  NNZ,Cy_CUSP.column_indices.begin());
   thrust::copy(Cy,Cy + NNZ,Cy_CUSP.values.begin());
   //---------------------------------------------- 
   
   //---------------------------------------------- 
   // Copy transpose(C_y) from host to device
   // Allocate stifness matrix transpose(C_y) in CSR format
   cusp::csr_matrix<int, real2, cusp::device_memory> CyT_CUSP(NN, NN, NNZ);
   thrust::copy(rowStartsSmall,rowStartsSmall + NN + 1,CyT_CUSP.row_offsets.begin());
   thrust::copy(colSmall,colSmall +  NNZ,CyT_CUSP.column_indices.begin());
   thrust::copy(CyT,CyT + NNZ,CyT_CUSP.values.begin());
   //---------------------------------------------- 
   End7 = getHighResolutionTime();    
   printf("         Time for copy Cy and CyT            = %-.4g seconds.\n", End7 - Start7);    
      
   Start7 = getHighResolutionTime();         
   //---------------------------------------------- 
   // Copy K_v^(-1) from host to device 
   // Allocate stifness matrix K_v^(-1) in CSR format   
   cusp::csr_matrix<int, real2, cusp::device_memory> K_v_diagonal_CUSP(NN, NN, NN);
   thrust::copy(rowStartsDiagonal,rowStartsDiagonal + NN + 1,K_v_diagonal_CUSP.row_offsets.begin());
   thrust::copy(colDiagonal,colDiagonal +  NN,K_v_diagonal_CUSP.column_indices.begin());
   thrust::copy(K_v_diagonal,K_v_diagonal + NN,K_v_diagonal_CUSP.values.begin()); 
   //----------------------------------------------     

   //---------------------------------------------- 
   // Copy velocities from host to device memory
   cusp::array1d<real2, cusp::device_memory> v_CUSP(NN);   
   thrust::copy(v, v + NN, v_CUSP.begin()); 
   //----------------------------------------------
   
   //----------------------------------------------  
   // RHS of the equation [4a]  
   // transpose(C_y)*u 
   // \______________/ 
   //        F2             
   cusp::array1d<real2, cusp::device_memory> F2(NN);
   cusp::multiply(CyT_CUSP, v_CUSP, F2);     
   cusp::blas::axpy(F2,Fsum,-1); 
   //---------------------------------------------- 
   
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      F2.swap(tmp);
   }    
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      v_CUSP.swap(tmp);
   }       
   End7 = getHighResolutionTime();    
   printf("         Time for sum [transpose(C_y)*v]     = %-.4g seconds.\n", End7 - Start7);
   
   Start7 = getHighResolutionTime();      
   //---------------------------------------------- 
   // LHS of the equation [4a]
   // transpose(C_y)*(diagonal(K_v)^-1
   cusp::csr_matrix<int, real2, cusp::device_memory> CyTdia;
   cusp::multiply(CyT_CUSP, K_v_diagonal_CUSP, CyTdia);
   //----------------------------------------------    
   
   {
      // create temporary empty matrix to delete array
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      CyT_CUSP.swap(tmp);
   } 
   {
      // create temporary empty matrix
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      K_v_diagonal_CUSP.swap(tmp);
   }
   End7 = getHighResolutionTime();   
   printf("         Time for [transpose(C_y)] * K_v     = %-.4g seconds.\n", End7 - Start7);   

   Start7 = getHighResolutionTime();   
   //----------------------------------------------   
   // LHS of the equation [4a]
   // [transpose(C_y)*(diagonal(K_v)^-1]*C_y 
   // \________________________________/
   //          from above (CyTdia)   
   cusp::csr_matrix<int, real2, cusp::device_memory> valy;   
   cusp::multiply(CyTdia, Cy_CUSP, valy);   
   // summing x, y components
   // [transpose(C_x)*(diagonal(K_u)^-1]*C_x + [transpose(C_y)*(diagonal(K_v)^-1]*C_y
   cusp::blas::axpy(valy.values,valx.values,1);
   //----------------------------------------------  
   
   {
      // create temporary empty matrix
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      valy.swap(tmp);
   }  
   End7 = getHighResolutionTime();   
   printf("         Time for [trans(C_y)*K_v] * C_y     = %-.4g seconds.\n", End7 - Start7);     
   End6 = getHighResolutionTime();   
   printf("      Time for calc pC arrays for y dim   = %-.4g seconds.\n", End6 - Start6);    
   

   Start6 = getHighResolutionTime();    
   //---------------------------------------------- 
   //calculate arrays for z dimension
   //---------------------------------------------- 
   
   Start7 = getHighResolutionTime();   
   //---------------------------------------------- 
   // Copy C_z from host to device
   // Allocate stifness matrix C_y in CSR format
   cusp::csr_matrix<int, real2, cusp::device_memory> Cz_CUSP(NN, NN, NNZ);
   thrust::copy(rowStartsSmall,rowStartsSmall + NN + 1,Cz_CUSP.row_offsets.begin());
   thrust::copy(colSmall,colSmall +  NNZ,Cz_CUSP.column_indices.begin());
   thrust::copy(Cz,Cz + NNZ,Cz_CUSP.values.begin());
   //---------------------------------------------- 
   
   //---------------------------------------------- 
   // Copy transpose(C_z) from host to device
   // Allocate stifness matrix transpose(C_z) in CSR format
   cusp::csr_matrix<int, real2, cusp::device_memory> CzT_CUSP(NN, NN, NNZ);
   thrust::copy(rowStartsSmall,rowStartsSmall + NN + 1,CzT_CUSP.row_offsets.begin());
   thrust::copy(colSmall,colSmall +  NNZ,CzT_CUSP.column_indices.begin());
   thrust::copy(CzT,CzT + NNZ,CzT_CUSP.values.begin());
   //---------------------------------------------- 
   End7 = getHighResolutionTime();    
   printf("         Time for copy Cz and CzT            = %-.4g seconds.\n", End7 - Start7);    
      
   Start7 = getHighResolutionTime();      
   //---------------------------------------------- 
   // Copy K_w^(-1) from host to device 
   // Allocate stifness matrix K_w^(-1) in CSR format   
   cusp::csr_matrix<int, real2, cusp::device_memory> K_w_diagonal_CUSP(NN, NN, NN);
   thrust::copy(rowStartsDiagonal,rowStartsDiagonal + NN + 1,K_w_diagonal_CUSP.row_offsets.begin());
   thrust::copy(colDiagonal,colDiagonal +  NN,K_w_diagonal_CUSP.column_indices.begin());
   thrust::copy(K_w_diagonal,K_w_diagonal + NN,K_w_diagonal_CUSP.values.begin()); 
   //----------------------------------------------     

   //---------------------------------------------- 
   // Copy velocities from host to device memory
   cusp::array1d<real2, cusp::device_memory> w_CUSP(NN);   
   thrust::copy(w, w + NN, w_CUSP.begin()); 
   //----------------------------------------------
   
   //----------------------------------------------  
   // RHS of the equation [4a]  
   // transpose(C_z)*u 
   // \______________/ 
   //        F2             
   cusp::array1d<real2, cusp::device_memory> F3(NN);
   cusp::multiply(CzT_CUSP, w_CUSP, F3);     
   cusp::blas::axpy(F3,Fsum,-1); 
   //---------------------------------------------- 
   
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      F3.swap(tmp);
   }    
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      w_CUSP.swap(tmp);
   }
   End7 = getHighResolutionTime();    
   printf("         Time for sum [transpose(C_z)*w]     = %-.4g seconds.\n", End7 - Start7);   

   Start7 = getHighResolutionTime();   
   //---------------------------------------------- 
   // LHS of the equation [4a]
   // transpose(C_z)*(diagonal(K_w)^-1
   cusp::csr_matrix<int, real2, cusp::device_memory> CzTdia;
   cusp::multiply(CzT_CUSP, K_w_diagonal_CUSP, CzTdia);
   //----------------------------------------------    
   
   {
      // create temporary empty matrix to delete array
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      CzT_CUSP.swap(tmp);
   } 
   {
      // create temporary empty matrix to delete array
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      K_w_diagonal_CUSP.swap(tmp);
   }   
   End7 = getHighResolutionTime();   
   printf("         Time for [transpose(C_z)] * K_w     = %-.4g seconds.\n", End7 - Start7);   

   Start7 = getHighResolutionTime();   
   //----------------------------------------------   
   // LHS of the equation [4a]
   // [transpose(C_z)*(diagonal(K_w)^-1]*C_z 
   // \________________________________/
   //          from above (CzTdia)   
   cusp::csr_matrix<int, real2, cusp::device_memory> valz;   
   cusp::multiply(CzTdia, Cz_CUSP, valz);   
   // summing x, y, z components
   // [transpose(C_x)*(diagonal(K_u)^-1]*C_x + [transpose(C_y)*(diagonal(K_v)^-1]*C_y + [transpose(C_z)*(diagonal(K_w)^-1]*C_z
   cusp::blas::axpy(valz.values,valx.values,1);
   //----------------------------------------------  
   
   {
      // create temporary empty matrix
      cusp::csr_matrix<int,real2,cusp::device_memory> tmp(1,1,1);
      valz.swap(tmp);
   }  
   End7 = getHighResolutionTime();   
   printf("         Time for [trans(C_z)*K_w] * C_z     = %-.4g seconds.\n", End7 - Start7);   
   End6 = getHighResolutionTime();   
   printf("      Time for calc pC arrays for z dim   = %-.4g seconds.\n", End6 - Start6); 
   
   
   Start6 = getHighResolutionTime();       
   // Copy resulting LHS and RHS vectors from device memory to host memory

   if (iter==1){   
      val_deltaP = new real2[valx.row_offsets[NN]];
      F_deltaP = new real2[NN];
      row_deltaP = new int[NN+1];
      col_deltaP = new int[valx.row_offsets[NN]];      
   }

   thrust::copy(valx.row_offsets.begin(), valx.row_offsets.end(), row_deltaP);
   thrust::copy(valx.column_indices.begin(), valx.column_indices.end(), col_deltaP);
   thrust::copy(valx.values.begin(), valx.values.end(), val_deltaP);
   
   thrust::copy(Fsum.begin(), Fsum.end(), F_deltaP);  
   {
      // create temporary empty matrix to delete array
      cusp::array1d<real2, cusp::device_memory> tmp(1);
      Fsum.swap(tmp);
   }
   applyBC_deltaP();

   thrust::copy(val_deltaP,val_deltaP + NNZ,valx.values.begin());   
   
   cusp::array1d<real2, cusp::device_memory> F(NN);
   thrust::copy(F_deltaP,F_deltaP + NN,F.begin());
   
   End6 = getHighResolutionTime();   
   printf("      Time for init variables for CR      = %-.4g seconds.\n", End6 - Start6);
   
   if (iter!=1) {   
      //----------------------------------------------
      //-------------CONJUGATE GRADIENT---------------

      Start6 = getHighResolutionTime();     
      //----------------------------------------------
      // Solve pressure correction equation [4a] with CUSP's CG

      cusp::array1d<real2, cusp::device_memory> x(NN);

      // Copy previous solution to device memory
      thrust::copy(delta_p, delta_p + NN, x.begin());
      
      // Set stopping criteria:
      //cusp::verbose_monitor<real2> monitor(b, solverIterMax, solverTol);
      cusp::default_monitor<real2> monitor(F, solverIterMax, solverTol);

      // Set preconditioner 
      // 1) identity
      //cusp::identity_operator<real2, cusp::device_memory> M(valx.num_rows, valx.num_rows);    
      // 2) smoothed aggregation preconditioner and jacobi smoother
      //cusp::precond::aggregation::smoothed_aggregation<int, real2, cusp::device_memory> M(valx);    
      // 3) smoothed aggregation preconditioner and polynomial smoother
	   //typedef cusp::relaxation::polynomial<int,cusp::device_memory> Smoother;      
      //cusp::precond::aggregation::smoothed_aggregation<int, real2, cusp::device_memory, Smoother> M(valx);    
      // 4) diagonal preconditioner
      cusp::precond::diagonal<real2, cusp::device_memory> M(valx);      
      
      // Solve the linear system A * x = Fsum with the Conjugate Gradient method
      // cusp::krylov::bicgstab(A, x, Fsum, monitor, M);
      //int restart = 40;
      // cout << "Iterative solution is started." << endl;
      cusp::krylov::cr(valx, x, F, monitor, M);
      // cout << "Iterative solution is finished." << endl;

      // Copy x from device back to u on host 
      thrust::copy(x.begin(), x.end(), delta_p);
      
      End6 = getHighResolutionTime();   
      printf("      Time for CR calculations            = %-.4g seconds.\n", End6 - Start6);     
      
      // report solver results
      if (monitor.converged())
      {
         std::cout << "      Solver converged to " << monitor.relative_tolerance() << " relative tolerance";
         std::cout << " after " << monitor.iteration_count() << " iterations";
         std::cout << " (" << monitor.residual_norm() << " final residual)" << endl;   
      }
      else
      {
         std::cout << "      Solver reached iteration limit " << monitor.iteration_limit() << " before converging";
         std::cout << " to " << monitor.relative_tolerance() << " relative tolerance " ;
         std::cout << " (" << monitor.residual_norm() << " final residual)" << endl;  
      }
   }

}  // End of function CUSP_pressureCorrection()
