#include "hip/hip_runtime.h"
#include <cusp/csr_matrix.h>
#include <cusp/print.h>
#include <cusp/krylov/cg.h>
#include <cusp/krylov/bicg.h>
#include <cusp/krylov/bicgstab.h>
#include <cusp/krylov/gmres.h>
//#include <cusp/precond/diagonal.h>
//#include <cusp/precond/ainv.h>
//#include <cusp/precond/smoothed_aggregation.h>

using namespace std;

#ifdef SINGLE
  typedef float real2;
#else
  typedef double real2;
#endif

extern int   *rowStarts, *col, Ndof, NNZ, solverIterMax, solverIter;
extern double solverTol, solverNorm;
extern real2 *u, *val, *F;
time_t start, end;


//-----------------------------------------------------------------------------
void CUSPsolver()
//-----------------------------------------------------------------------------
{
   // Solve system of linear equations using an iterative solver of CUSP on a GPU.

   //cout << endl << "Start of CUSPsolver() function." << endl;
   //cout << "Ndof = " << Ndof << endl;
   //cout << "NNZ  = " << NNZ << endl;

   // Allocate stifness matrix [A] in CSR format and right hand side vector {b}
   // and solution vector {x} in device memory.
   cusp::csr_matrix<int, real2, cusp::device_memory> A(Ndof, Ndof, NNZ);
   cusp::array1d<real2, cusp::device_memory> b(Ndof);
   cusp::array1d<real2, cusp::device_memory> x(Ndof);

   // Copy CSR row pointers to device memory
   thrust::copy(rowStarts, rowStarts + Ndof + 1, A.row_offsets.begin());

   // Copy CSR column indices to device memory
   thrust::copy(col, col +  NNZ, A.column_indices.begin());

   // Copy CSR values to device memory
   thrust::copy(val, val + NNZ, A.values.begin()); 

   // Copy right hand side vector to device memory
   thrust::copy(F, F + Ndof, b.begin());

   // Copy previous solution to device memory
   thrust::copy(u, u + Ndof, x.begin());
   
   //cusp::verbose_monitor<real2> monitor(b, solverIterMax, solverTol);
   cusp::default_monitor<real2> monitor(b, solverIterMax, solverTol);

   // Set preconditioner
   //cusp::identity_operator<real2, cusp::device_memory> M(A.num_rows, A.num_rows);
   //cusp::precond::diagonal<real2, cusp::device_memory> M(A);
   //cusp::precond::scaled_bridson_ainv<real2, cusp::device_memory> M(A, .1);
   //cusp::precond::smoothed_aggregation<int, real2, cusp::device_memory> M(A);

   int restart = 20;
   
   //cout << "   GMRES solver is started... " << endl;
   //time (&start);
   cusp::krylov::gmres(A, x, b, restart, monitor);
   //time (&end);
   //cout << "   Done. Elapsed wall clock time is " << difftime (end,start) << " seconds." << endl;

   // Copy x from device back to u on host 
   thrust::copy(x.begin(), x.end(), u);
   
   solverIter = monitor.iteration_count();
   solverNorm = monitor.residual_norm();

   // ----------------------CONTROL------------------------
   // Print the solution to check
   // for(int i=0; i<=Ndof; i++) {
   //    cout << "u[" << i << "] = " << u[i] << endl;
   // }
   // ----------------------CONTROL------------------------

   //cout << "End of CUSPsolver() function." << endl;

}  // End of function CUSPSolver()


